#include "hip/hip_runtime.h"
#include "precompute_interface.h"
#include "precompute_engine_cuda.h" // For launching the kernel and physics
#include "cuda_host_utils.h"      // For Healpix utilities on host (still needed for initial unique IDs)
#include "healpix_gpu.h"          // For GPU-side HEALPix functions
// Replace broken kdtree with cudaKDTree
#include "../cudaKDTree/cukd/builder.h"
#include "../cudaKDTree/cukd/knn.h"
#include <cfloat>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>       // For float3
#include <vector>
#include <iostream> // For error messages / cout
#include <algorithm> // For std::sort, std::unique, std::max
#include <map>       // For std::map (can be removed if not used for unique_pix anymore)
#include <cmath>     // For sqrt, M_PI
#include <stdexcept> // For std::runtime_error

// Ensure M_PI is defined
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// DEG2RAD, if still needed (likely replaced by direct radian usage)
#ifndef DEG2RAD_Interface
#define DEG2RAD_Interface 0.017453292519943295
#endif

// Constants for HEALPix - Host version
#ifndef TWO_PI
#define TWO_PI (2.0 * M_PI)
#endif

#define TWOTHIRD (2.0 / 3.0)

// Host version of acos_safe function
double acos_safe_host(double val) {
    if (val <= -1.0) {
        return M_PI;
    } else if (val >= 1.0) {
        return 0.0;
    } else {
        return acos(val);
    }
}

// Host version of pix2ang_ring function (copied from healpix_gpu.cu device version)
void pix2ang_ring_host(long nside, long pix, double& theta, double& phi) {
    if (nside <= 0 || pix < 0 || pix >= 12 * nside * nside) {
        theta = phi = 0.0;
        return;
    }

    const long npix = 12 * nside * nside;
    const long ncap = 2 * nside * (nside - 1);
    const long nsidesq = nside * nside;
    const long nl2 = 2 * nside;
    const long nl4 = 4 * nside;

    long ipix1 = pix + 1; // Convert to 1-based indexing as in Java

    if (ipix1 <= ncap) { // North polar cap
        double hip = ipix1 / 2.0;
        double fihip = (long) hip; // get integer part of hip
        long iring = (long) (sqrt(hip - sqrt(fihip))) + 1; // counted from north pole
        long iphi = ipix1 - 2 * iring * (iring - 1);
        
        theta = acos_safe_host(1.0 - iring * iring / (3.0 * nsidesq));
        phi = ((double)iphi - 0.5) * M_PI / (2.0 * iring);
        
    } else if (ipix1 <= nl2 * (5 * nside + 1)) { // Equatorial region
        long ip = ipix1 - ncap - 1;
        long iring = (ip / nl4) + nside; // counted from North pole
        long iphi = ip % nl4 + 1;
        
        double fodd = 0.5 * (1.0 + ((iring + nside) % 2)); // 1 if iring+nside is odd, 1/2 otherwise
        
        theta = acos_safe_host((nl2 - iring) / (1.5 * nside));
        phi = ((double)iphi - fodd) * M_PI / (2.0 * nside);

    } else { // South polar cap
        long ip = npix - ipix1 + 1;
        double hip = ip / 2.0;
        double fihip = (long) hip;
        long iring = (long) (sqrt(hip - sqrt(fihip))) + 1; // counted from South pole
        long iphi = 4 * iring + 1 - (ip - 2 * iring * (iring - 1));
        
        theta = acos_safe_host(-1.0 + iring * iring / (3.0 * nsidesq));
        phi = ((double)iphi - 0.5) * M_PI / (2.0 * iring);
    }
    
    // Normalize phi to [0, 2*PI)
    while (phi < 0.0) phi += TWO_PI;
    while (phi >= TWO_PI) phi -= TWO_PI;
}

// Helper macro for CUDA error checking
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err_) << " in file " << __FILE__ << " at line " << __LINE__ << std::endl; \
        return -1; \
    } \
}

// Structure for passing data to the source processing
typedef struct {
    int lens_idx;
    int N_bins;
    long nside_healpix;

    // Source data (global arrays)
    const long* g_unique_source_hp_ids;
    const long* g_all_source_hp_ids_sorted;
    const int* g_sorted_source_original_indices;
    const int* g_unique_hp_id_offsets_start;
    const int* g_unique_hp_id_offsets_end;

    // Lens data
    double lens_zl_i;
    double lens_dcoml_i;
    double lens_sin_ra_l_i;
    double lens_cos_ra_l_i;
    double lens_sin_dec_l_i;
    double lens_cos_dec_l_i;

    // Pointers to global source property arrays
    const double* g_z_s;
    const double* g_d_com_s;
    const double* g_sin_ra_s;
    const double* g_cos_ra_s;
    const double* g_sin_dec_s;
    const double* g_cos_dec_s;
    const double* g_w_s;
    const double* g_e_1_s;
    const double* g_e_2_s;
    const double* g_z_l_max_s;

    bool has_sigma_crit_eff;
    int n_z_bins_l;
    const double* g_sigma_crit_eff_l;
    const int* g_z_bin_s;

    bool has_m_s; const double* g_m_s;
    bool has_e_rms_s; const double* g_e_rms_s;
    bool has_R_2_s; const double* g_R_2_s;
    bool has_R_matrix_s;
    const double* g_R_11_s; const double* g_R_12_s;
    const double* g_R_21_s; const double* g_R_22_s;

    const double* g_dist_3d_sq_bins;

    // Configuration
    bool comoving;
    int weighting;

    // Output sum arrays (global pointers)
    long long* g_sum_1_r;
    double* g_sum_w_ls_r;
    double* g_sum_w_ls_e_t_r;
    double* g_sum_w_ls_e_t_sigma_crit_r;
    double* g_sum_w_ls_z_s_r;
    double* g_sum_w_ls_sigma_crit_r;
    double* g_sum_w_ls_m_r;
    double* g_sum_w_ls_1_minus_e_rms_sq_r;
    double* g_sum_w_ls_A_p_R_2_r;
    double* g_sum_w_ls_R_T_r;

} KernelCallbackData;

// Device function to process a single source HP pixel that was found by KNN search
__device__ void process_found_source_hp_pixel(int source_kdtree_idx, KernelCallbackData* cb_data) {
    // 1. Get the unique HEALPix ID for the found KD-tree node
    long unique_hp_id = cb_data->g_unique_source_hp_ids[source_kdtree_idx];

    // 2. Find the range of actual sources belonging to this unique_hp_id
    int start_offset = cb_data->g_unique_hp_id_offsets_start[source_kdtree_idx];
    int end_offset = cb_data->g_unique_hp_id_offsets_end[source_kdtree_idx];
    
    // Debug output can be enabled by uncommenting:
    // if (cb_data->lens_idx < 3 && source_kdtree_idx < 3) {
    //     printf("  Processing HP pixel %d: hp_id=%ld, sources [%d:%d)\n", 
    //            source_kdtree_idx, unique_hp_id, start_offset, end_offset);
    // }

    // 3. Loop over these actual source indices
    for (int i_s_mapped_idx = start_offset; i_s_mapped_idx < end_offset; ++i_s_mapped_idx) {
        int original_source_idx = cb_data->g_sorted_source_original_indices[i_s_mapped_idx];

        // Load current source data using original_source_idx
        double zs_i = cb_data->g_z_s[original_source_idx];

        // Redshift filter
        if (cb_data->lens_zl_i >= zs_i || cb_data->lens_zl_i >= cb_data->g_z_l_max_s[original_source_idx]) {
            continue;
        }

        double dcoms_i = cb_data->g_d_com_s[original_source_idx];
        double sin_ra_s_i = cb_data->g_sin_ra_s[original_source_idx];
        double cos_ra_s_i = cb_data->g_cos_ra_s[original_source_idx];
        double sin_dec_s_i = cb_data->g_sin_dec_s[original_source_idx];
        double cos_dec_s_i = cb_data->g_cos_dec_s[original_source_idx];

        // Calculate 3D angular distance squared to match CPU implementation
        // The CPU uses dist_3d_sq which is 3D chord distance on unit sphere
        double dist_sq = dist_angular_sq_gpu(
            cb_data->lens_sin_ra_l_i, cb_data->lens_cos_ra_l_i,
            cb_data->lens_sin_dec_l_i, cb_data->lens_cos_dec_l_i,
            sin_ra_s_i, cos_ra_s_i, sin_dec_s_i, cos_dec_s_i
        );

        // Find bin index
        const double* current_lens_dist_bins = cb_data->g_dist_3d_sq_bins + (size_t)cb_data->lens_idx * (cb_data->N_bins + 1);
        int i_bin = find_bin_idx_gpu(dist_sq, current_lens_dist_bins, cb_data->N_bins);

        if (i_bin == -1) { // Not in any bin for this lens
            continue;
        }

        // Calculate sigma_crit_inverse
        int z_bin_s_val = -1;
        if (cb_data->has_sigma_crit_eff && cb_data->g_z_bin_s != nullptr) {
            z_bin_s_val = cb_data->g_z_bin_s[original_source_idx];
        }
        double sigma_crit_inv = calculate_sigma_crit_inv_gpu(
            cb_data->lens_zl_i, zs_i, cb_data->lens_dcoml_i, dcoms_i,
            cb_data->comoving,
            cb_data->has_sigma_crit_eff,
            cb_data->g_sigma_crit_eff_l,
            cb_data->lens_idx,
            cb_data->n_z_bins_l,
            z_bin_s_val
        );

        if (sigma_crit_inv == 0.0 || sigma_crit_inv == DBL_MAX) {
            continue;
        }

        // Calculate w_ls
        double w_ls = calculate_w_ls_gpu(
            sigma_crit_inv,
            cb_data->g_w_s[original_source_idx],
            cb_data->weighting
        );

        if (w_ls == 0.0) {
            continue;
        }

        // Calculate tangential shear e_t
        double cos_2phi, sin_2phi;
        calculate_et_components_gpu(
            cb_data->lens_sin_ra_l_i, cb_data->lens_cos_ra_l_i,
            cb_data->lens_sin_dec_l_i, cb_data->lens_cos_dec_l_i,
            sin_ra_s_i, cos_ra_s_i, sin_dec_s_i, cos_dec_s_i,
            cos_2phi, sin_2phi
        );
        double e_t_val = calculate_et_gpu(
            cb_data->g_e_1_s[original_source_idx],
            cb_data->g_e_2_s[original_source_idx],
            cos_2phi, sin_2phi
        );

        // Accumulate results using atomics
        size_t out_idx = (size_t)cb_data->lens_idx * cb_data->N_bins + i_bin;

        atomicAdd((unsigned long long int*)&cb_data->g_sum_1_r[out_idx], 1ULL);
        atomicAdd(&cb_data->g_sum_w_ls_r[out_idx], w_ls);
        atomicAdd(&cb_data->g_sum_w_ls_e_t_r[out_idx], w_ls * e_t_val);
        
        double sigma_crit = (sigma_crit_inv == 0.0) ? DBL_MAX : 1.0 / sigma_crit_inv;
        atomicAdd(&cb_data->g_sum_w_ls_e_t_sigma_crit_r[out_idx], w_ls * e_t_val * sigma_crit);
        atomicAdd(&cb_data->g_sum_w_ls_z_s_r[out_idx], w_ls * zs_i);
        atomicAdd(&cb_data->g_sum_w_ls_sigma_crit_r[out_idx], w_ls * sigma_crit);

        if (cb_data->has_m_s && cb_data->g_m_s != nullptr && cb_data->g_sum_w_ls_m_r != nullptr) {
            atomicAdd(&cb_data->g_sum_w_ls_m_r[out_idx], w_ls * cb_data->g_m_s[original_source_idx]);
        }
        if (cb_data->has_e_rms_s && cb_data->g_e_rms_s != nullptr && cb_data->g_sum_w_ls_1_minus_e_rms_sq_r != nullptr) {
            double e_rms_s_i = cb_data->g_e_rms_s[original_source_idx];
            atomicAdd(&cb_data->g_sum_w_ls_1_minus_e_rms_sq_r[out_idx], w_ls * (1.0 - e_rms_s_i * e_rms_s_i));
        }
        if (cb_data->has_R_2_s && cb_data->g_R_2_s != nullptr && cb_data->g_sum_w_ls_A_p_R_2_r != nullptr) {
            if (cb_data->g_R_2_s[original_source_idx] <= 0.31) {
                // Match CPU implementation: 0.00865 * w_ls / 0.01 = 0.865 * w_ls
                atomicAdd(&cb_data->g_sum_w_ls_A_p_R_2_r[out_idx], 0.00865 * w_ls / 0.01);
            }
        }
        if (cb_data->has_R_matrix_s && cb_data->g_R_11_s != nullptr && cb_data->g_sum_w_ls_R_T_r != nullptr) {
            double R_T_val = calculate_R_T_gpu(
                cb_data->g_R_11_s[original_source_idx], cb_data->g_R_12_s[original_source_idx],
                cb_data->g_R_21_s[original_source_idx], cb_data->g_R_22_s[original_source_idx],
                cos_2phi, sin_2phi
            );
            atomicAdd(&cb_data->g_sum_w_ls_R_T_r[out_idx], w_ls * R_T_val);
        }
    }
}

__global__ void process_all_lenses_kernel(
    // Lens data (global arrays)
    const double* g_z_l, const double* g_d_com_l,
    const double* g_sin_ra_l, const double* g_cos_ra_l,
    const double* g_sin_dec_l, const double* g_cos_dec_l,
    const double* g_dist_3d_sq_bins,

    // Source data (global arrays)
    const double* g_z_s, const double* g_d_com_s,
    const double* g_sin_ra_s, const double* g_cos_ra_s,
    const double* g_sin_dec_s, const double* g_cos_dec_s,
    const double* g_w_s, const double* g_e_1_s, const double* g_e_2_s,
    const double* g_z_l_max_s,

    // Unique Source HEALPix data for KD-Tree
    const float3* g_unique_source_hp_coords_kdtree,
    const long* g_unique_source_hp_ids,
    int N_unique_source_hp,

    // Mapping from unique source HP ID back to original source indices
    const long* g_all_source_hp_ids_sorted,
    const int* g_sorted_source_original_indices,
    const int* g_unique_hp_id_offsets_start,
    const int* g_unique_hp_id_offsets_end,

    // Optional source data
    bool _has_sigma_crit_eff, int _n_z_bins_l, const double* _g_sigma_crit_eff_l, const int* _g_z_bin_s,
    bool _has_m_s, const double* _g_m_s,
    bool _has_e_rms_s, const double* _g_e_rms_s,
    bool _has_R_2_s, const double* _g_R_2_s,
    bool _has_R_matrix_s, const double* _g_R_11_s, const double* _g_R_12_s, const double* _g_R_21_s, const double* _g_R_22_s,

    // Configuration
    int N_lenses, int N_bins, long nside_healpix, bool comoving, int weighting,

    // Output sum arrays
    long long* g_sum_1_r, double* g_sum_w_ls_r,
    double* g_sum_w_ls_e_t_r, double* g_sum_w_ls_e_t_sigma_crit_r,
    double* g_sum_w_ls_z_s_r, double* g_sum_w_ls_sigma_crit_r,
    double* g_sum_w_ls_m_r, double* g_sum_w_ls_1_minus_e_rms_sq_r,
    double* g_sum_w_ls_A_p_R_2_r, double* g_sum_w_ls_R_T_r
) {
    int lens_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (lens_idx >= N_lenses) return;

    // Load current lens data
    double zl_i = g_z_l[lens_idx];
    double dcoml_i = g_d_com_l[lens_idx];
    double sin_ra_l_i = g_sin_ra_l[lens_idx];
    double cos_ra_l_i = g_cos_ra_l[lens_idx];
    double sin_dec_l_i = g_sin_dec_l[lens_idx];
    double cos_dec_l_i = g_cos_dec_l[lens_idx];

    float3 lens_xyz_cartesian;
    spherical_to_cartesian_gpu(sin_ra_l_i, cos_ra_l_i, sin_dec_l_i, cos_dec_l_i, lens_xyz_cartesian);

    // Calculate search radius for KNN to match CPU implementation
    // The CPU uses dist_3d_sq_bins which are 3D angular distances squared on unit sphere
    double max_dist_3d_sq_lens = g_dist_3d_sq_bins[lens_idx * (N_bins + 1) + N_bins];
    double lens_hp_pixrad = get_max_pixrad_gpu(nside_healpix);
    
    // Convert HEALPix pixel radius from radians to 3D chord distance
    double pixrad_3d_sq = 4.0 * sin(lens_hp_pixrad * 0.5) * sin(lens_hp_pixrad * 0.5);
    
    // Add buffer for pixel radius (matching CPU implementation)
    double search_radius_sq_gpu = max_dist_3d_sq_lens + 
                                  (4.0 * pixrad_3d_sq + 
                                   4.0 * sqrt(max_dist_3d_sq_lens) * sqrt(pixrad_3d_sq));
    
    float search_radius_sq = (float)search_radius_sq_gpu;
    
    // Debug output can be enabled by uncommenting:
    // if (lens_idx < 3) {
    //     printf("Lens %d: max_dist_3d_sq=%.6e, pixrad=%.6e, search_radius_sq=%.6e\n", 
    //            lens_idx, max_dist_3d_sq_lens, lens_hp_pixrad, search_radius_sq_gpu);
    // }

    // Prepare callback data
    KernelCallbackData callback_data;
    callback_data.lens_idx = lens_idx;
    callback_data.N_bins = N_bins;
    callback_data.nside_healpix = nside_healpix;
    callback_data.g_unique_source_hp_ids = g_unique_source_hp_ids;
    callback_data.g_all_source_hp_ids_sorted = g_all_source_hp_ids_sorted;
    callback_data.g_sorted_source_original_indices = g_sorted_source_original_indices;
    callback_data.g_unique_hp_id_offsets_start = g_unique_hp_id_offsets_start;
    callback_data.g_unique_hp_id_offsets_end = g_unique_hp_id_offsets_end;
    callback_data.lens_zl_i = zl_i;
    callback_data.lens_dcoml_i = dcoml_i;
    callback_data.lens_sin_ra_l_i = sin_ra_l_i;
    callback_data.lens_cos_ra_l_i = cos_ra_l_i;
    callback_data.lens_sin_dec_l_i = sin_dec_l_i;
    callback_data.lens_cos_dec_l_i = cos_dec_l_i;
    callback_data.g_z_s = g_z_s;
    callback_data.g_d_com_s = g_d_com_s;
    callback_data.g_sin_ra_s = g_sin_ra_s;
    callback_data.g_cos_ra_s = g_cos_ra_s;
    callback_data.g_sin_dec_s = g_sin_dec_s;
    callback_data.g_cos_dec_s = g_cos_dec_s;
    callback_data.g_w_s = g_w_s;
    callback_data.g_e_1_s = g_e_1_s;
    callback_data.g_e_2_s = g_e_2_s;
    callback_data.g_z_l_max_s = g_z_l_max_s;
    callback_data.has_sigma_crit_eff = _has_sigma_crit_eff;
    callback_data.n_z_bins_l = _n_z_bins_l;
    callback_data.g_sigma_crit_eff_l = _g_sigma_crit_eff_l;
    callback_data.g_z_bin_s = _g_z_bin_s;
    callback_data.has_m_s = _has_m_s; callback_data.g_m_s = _g_m_s;
    callback_data.has_e_rms_s = _has_e_rms_s; callback_data.g_e_rms_s = _g_e_rms_s;
    callback_data.has_R_2_s = _has_R_2_s; callback_data.g_R_2_s = _g_R_2_s;
    callback_data.has_R_matrix_s = _has_R_matrix_s;
    callback_data.g_R_11_s = _g_R_11_s; callback_data.g_R_12_s = _g_R_12_s;
    callback_data.g_R_21_s = _g_R_21_s; callback_data.g_R_22_s = _g_R_22_s;
    callback_data.g_dist_3d_sq_bins = g_dist_3d_sq_bins;
    callback_data.comoving = comoving;
    callback_data.weighting = weighting;
    callback_data.g_sum_1_r = g_sum_1_r;
    callback_data.g_sum_w_ls_r = g_sum_w_ls_r;
    callback_data.g_sum_w_ls_e_t_r = g_sum_w_ls_e_t_r;
    callback_data.g_sum_w_ls_e_t_sigma_crit_r = g_sum_w_ls_e_t_sigma_crit_r;
    callback_data.g_sum_w_ls_z_s_r = g_sum_w_ls_z_s_r;
    callback_data.g_sum_w_ls_sigma_crit_r = g_sum_w_ls_sigma_crit_r;
    callback_data.g_sum_w_ls_m_r = g_sum_w_ls_m_r;
    callback_data.g_sum_w_ls_1_minus_e_rms_sq_r = g_sum_w_ls_1_minus_e_rms_sq_r;
    callback_data.g_sum_w_ls_A_p_R_2_r = g_sum_w_ls_A_p_R_2_r;
    callback_data.g_sum_w_ls_R_T_r = g_sum_w_ls_R_T_r;

    // Use simple brute force radius search since kdtree_search_gpu.cu is broken
    // Loop through all unique source HEALPix pixels and check distance
    int candidates_found = 0;
    
    for (int source_kdtree_idx = 0; source_kdtree_idx < N_unique_source_hp; source_kdtree_idx++) {
        // Calculate distance squared between lens and this source HEALPix pixel
        float3 source_xyz = g_unique_source_hp_coords_kdtree[source_kdtree_idx];
        
        float dx = lens_xyz_cartesian.x - source_xyz.x;
        float dy = lens_xyz_cartesian.y - source_xyz.y;
        float dz = lens_xyz_cartesian.z - source_xyz.z;
        float dist_sq = dx * dx + dy * dy + dz * dz;
        
        // If within search radius, process this HEALPix pixel
        if (dist_sq <= search_radius_sq) {
            candidates_found++;
            process_found_source_hp_pixel(source_kdtree_idx, &callback_data);
        }
    }
    
    // Debug output can be enabled by uncommenting:
    // if (lens_idx < 5) {
    //     printf("Lens %d: search_radius_sq=%.6e, candidates_found=%d\n", 
    //            lens_idx, search_radius_sq, candidates_found);
    // }
}

int precompute_cuda_interface(TableData* tables, int n_gpus) {
    // --- 1. Input Validation (remains similar) ---
    if (!tables->z_l || !tables->d_com_l || !tables->sin_ra_l || !tables->cos_ra_l || !tables->sin_dec_l || !tables->cos_dec_l ||
        !tables->z_s || !tables->d_com_s || !tables->sin_ra_s || !tables->cos_ra_s || !tables->sin_dec_s || !tables->cos_dec_s || !tables->w_s || !tables->e_1_s || !tables->e_2_s || !tables->z_l_max_s || !tables->healpix_id_s ||
        !tables->dist_3d_sq_bins ||
        !tables->sum_1_r || !tables->sum_w_ls_r || !tables->sum_w_ls_e_t_r || !tables->sum_w_ls_e_t_sigma_crit_r || !tables->sum_w_ls_z_s_r || !tables->sum_w_ls_sigma_crit_r) {
        std::cerr << "Error: Essential data pointers in TableData are null." << std::endl;
        return -1;
    }
     if (tables->has_sigma_crit_eff && (!tables->sigma_crit_eff_l || !tables->z_bin_s)) {
        std::cerr << "Error: sigma_crit_eff pointers are null when has_sigma_crit_eff is true." << std::endl;
        return -1;
    }
    if (tables->has_m_s && !tables->m_s) { std::cerr << "Error: m_s is null when has_m_s is true." << std::endl; return -1; }
    // ... other validation checks ...
    if (tables->n_lenses <= 0 && tables->n_sources == 0) {
        std::cout << "No lenses and no sources. Exiting." << std::endl;
        return 0; // No work to do
    }
     if (tables->n_lenses > 0 && tables->n_bins <= 0) {
        std::cerr << "Error: n_bins must be positive if there are lenses." << std::endl;
        return -1;
    }
    if (tables->nside_healpix <= 0 && (tables->n_lenses > 0 || tables->n_sources > 0)) {
        std::cerr << "Error: nside_healpix must be positive if there are objects." << std::endl;
        return -1;
    }


    // --- 2. GPU Setup ---
    CUDA_CHECK(hipSetDevice(0)); // Use GPU 0 by default

    // --- 3. GPU Data Allocation and H2D Transfers (Upfront) ---
    // Device pointers
    double *d_z_l, *d_d_com_l, *d_sin_ra_l, *d_cos_ra_l, *d_sin_dec_l, *d_cos_dec_l;
    double *d_dist_3d_sq_bins;

    double *d_z_s, *d_d_com_s, *d_sin_ra_s, *d_cos_ra_s, *d_sin_dec_s, *d_cos_dec_s;
    double *d_w_s, *d_e_1_s, *d_e_2_s, *d_z_l_max_s;
    long* d_all_source_hp_ids; // All source HP IDs, unsorted initially

    double *d_sigma_crit_eff_l = nullptr; int *d_z_bin_s = nullptr;
    double *d_m_s = nullptr, *d_e_rms_s = nullptr, *d_R_2_s = nullptr;
    double *d_R_11_s = nullptr, *d_R_12_s = nullptr, *d_R_21_s = nullptr, *d_R_22_s = nullptr;

    // Allocate and copy lens data
    CUDA_CHECK(hipMalloc(&d_z_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_z_l, tables->z_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_d_com_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_d_com_l, tables->d_com_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));
    // ... (repeat for sin_ra_l, cos_ra_l, sin_dec_l, cos_dec_l) ...
    CUDA_CHECK(hipMalloc(&d_sin_ra_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_sin_ra_l, tables->sin_ra_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_cos_ra_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_cos_ra_l, tables->cos_ra_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_sin_dec_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_sin_dec_l, tables->sin_dec_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_cos_dec_l, tables->n_lenses * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_cos_dec_l, tables->cos_dec_l, tables->n_lenses * sizeof(double), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_dist_3d_sq_bins, (size_t)tables->n_lenses * (tables->n_bins + 1) * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_dist_3d_sq_bins, tables->dist_3d_sq_bins, (size_t)tables->n_lenses * (tables->n_bins + 1) * sizeof(double), hipMemcpyHostToDevice));

    // Allocate and copy source data
    if (tables->n_sources > 0) {
        CUDA_CHECK(hipMalloc(&d_z_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_z_s, tables->z_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        // ... (repeat for all source arrays: d_com_s, sin/cos_ra/dec_s, w_s, e1/2_s, z_l_max_s, healpix_id_s) ...
        CUDA_CHECK(hipMalloc(&d_d_com_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_d_com_s, tables->d_com_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_sin_ra_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_sin_ra_s, tables->sin_ra_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_cos_ra_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_cos_ra_s, tables->cos_ra_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_sin_dec_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_sin_dec_s, tables->sin_dec_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_cos_dec_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_cos_dec_s, tables->cos_dec_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_w_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_w_s, tables->w_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_e_1_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_e_1_s, tables->e_1_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_e_2_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_e_2_s, tables->e_2_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_z_l_max_s, tables->n_sources * sizeof(double)));
        CUDA_CHECK(hipMemcpy(d_z_l_max_s, tables->z_l_max_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_all_source_hp_ids, tables->n_sources * sizeof(long)));
        CUDA_CHECK(hipMemcpy(d_all_source_hp_ids, tables->healpix_id_s, tables->n_sources * sizeof(long), hipMemcpyHostToDevice));


        if (tables->has_sigma_crit_eff) {
            CUDA_CHECK(hipMalloc(&d_sigma_crit_eff_l, (size_t)tables->n_lenses * tables->n_z_bins_l * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_sigma_crit_eff_l, tables->sigma_crit_eff_l, (size_t)tables->n_lenses * tables->n_z_bins_l * sizeof(double), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_z_bin_s, tables->n_sources * sizeof(int)));
            CUDA_CHECK(hipMemcpy(d_z_bin_s, tables->z_bin_s, tables->n_sources * sizeof(int), hipMemcpyHostToDevice));
        }
        // ... (optional source arrays m_s, e_rms_s, etc.) ...
        if (tables->has_m_s) {
            CUDA_CHECK(hipMalloc(&d_m_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_m_s, tables->m_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        }
         if (tables->has_e_rms_s) {
            CUDA_CHECK(hipMalloc(&d_e_rms_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_e_rms_s, tables->e_rms_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        }
        if (tables->has_R_2_s) {
            CUDA_CHECK(hipMalloc(&d_R_2_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_R_2_s, tables->R_2_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        }
        if (tables->has_R_matrix_s) {
            CUDA_CHECK(hipMalloc(&d_R_11_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_R_11_s, tables->R_11_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_R_12_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_R_12_s, tables->R_12_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_R_21_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_R_21_s, tables->R_21_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_R_22_s, tables->n_sources * sizeof(double)));
            CUDA_CHECK(hipMemcpy(d_R_22_s, tables->R_22_s, tables->n_sources * sizeof(double), hipMemcpyHostToDevice));
        }
    } else { // n_sources == 0
        // Set source data pointers to null if no sources
        d_z_s = d_d_com_s = d_sin_ra_s = d_cos_ra_s = d_sin_dec_s = d_cos_dec_s = nullptr;
        d_w_s = d_e_1_s = d_e_2_s = d_z_l_max_s = nullptr;
        d_all_source_hp_ids = nullptr;
        // Optional ones too
        d_sigma_crit_eff_l = nullptr; d_z_bin_s = nullptr; // Note: d_sigma_crit_eff_l is lens-based but used with z_bin_s
        d_m_s = d_e_rms_s = d_R_2_s = nullptr;
        d_R_11_s = d_R_12_s = d_R_21_s = d_R_22_s = nullptr;
    }


    // Allocate and initialize output sum arrays on GPU
    size_t total_output_bins = (size_t)tables->n_lenses * tables->n_bins;
    long long* d_sum_1_r;
    double *d_sum_w_ls_r, *d_sum_w_ls_e_t_r, *d_sum_w_ls_e_t_sigma_crit_r;
    double *d_sum_w_ls_z_s_r, *d_sum_w_ls_sigma_crit_r;
    double *d_sum_w_ls_m_r = nullptr, *d_sum_w_ls_1_minus_e_rms_sq_r = nullptr;
    double *d_sum_w_ls_A_p_R_2_r = nullptr, *d_sum_w_ls_R_T_r = nullptr;

    if (total_output_bins > 0) {
        CUDA_CHECK(hipMalloc(&d_sum_1_r, total_output_bins * sizeof(long long)));
        CUDA_CHECK(hipMemset(d_sum_1_r, 0, total_output_bins * sizeof(long long)));
        CUDA_CHECK(hipMalloc(&d_sum_w_ls_r, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMemset(d_sum_w_ls_r, 0, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_sum_w_ls_e_t_r, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMemset(d_sum_w_ls_e_t_r, 0, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_sum_w_ls_e_t_sigma_crit_r, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMemset(d_sum_w_ls_e_t_sigma_crit_r, 0, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_sum_w_ls_z_s_r, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMemset(d_sum_w_ls_z_s_r, 0, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_sum_w_ls_sigma_crit_r, total_output_bins * sizeof(double)));
        CUDA_CHECK(hipMemset(d_sum_w_ls_sigma_crit_r, 0, total_output_bins * sizeof(double)));

        if (tables->has_m_s && tables->sum_w_ls_m_r) {
            CUDA_CHECK(hipMalloc(&d_sum_w_ls_m_r, total_output_bins * sizeof(double)));
            CUDA_CHECK(hipMemset(d_sum_w_ls_m_r, 0, total_output_bins * sizeof(double)));
        }
        if (tables->has_e_rms_s && tables->sum_w_ls_1_minus_e_rms_sq_r) {
            CUDA_CHECK(hipMalloc(&d_sum_w_ls_1_minus_e_rms_sq_r, total_output_bins * sizeof(double)));
            CUDA_CHECK(hipMemset(d_sum_w_ls_1_minus_e_rms_sq_r, 0, total_output_bins * sizeof(double)));
        }
        if (tables->has_R_2_s && tables->sum_w_ls_A_p_R_2_r) {
            CUDA_CHECK(hipMalloc(&d_sum_w_ls_A_p_R_2_r, total_output_bins * sizeof(double)));
            CUDA_CHECK(hipMemset(d_sum_w_ls_A_p_R_2_r, 0, total_output_bins * sizeof(double)));
        }
        if (tables->has_R_matrix_s && tables->sum_w_ls_R_T_r) {
            CUDA_CHECK(hipMalloc(&d_sum_w_ls_R_T_r, total_output_bins * sizeof(double)));
            CUDA_CHECK(hipMemset(d_sum_w_ls_R_T_r, 0, total_output_bins * sizeof(double)));
        }
    } else {
      if (tables->n_lenses == 0) {
        std::cout << "No lenses to process. Exiting early." << std::endl;
        if (d_all_source_hp_ids) CUDA_CHECK(hipFree(d_all_source_hp_ids));
        if (d_z_s) CUDA_CHECK(hipFree(d_z_s)); if (d_d_com_s) CUDA_CHECK(hipFree(d_d_com_s));
        return 0;
      }
    }

    // --- 4. GPU-Side Data Preparation for KD-Tree ---
    float3* d_unique_source_hp_coords_kdtree = nullptr;
    long* d_unique_source_hp_ids = nullptr;
    int N_unique_source_hp = 0;

    // For mapping unique HP IDs to original source indices ranges
    long* d_all_source_hp_ids_sorted_gpu = nullptr;
    int* d_sorted_source_original_indices_gpu = nullptr;
    int* d_unique_hp_id_offsets_start_gpu = nullptr;
    int* d_unique_hp_id_offsets_end_gpu = nullptr;


    if (tables->n_sources > 0) {
        // --- Conceptual Steps for KD-Tree setup ---
        // 1. Create a temporary array of (hp_id, original_idx) pairs on host or device.
        std::vector<std::pair<long, int>> source_hp_pairs(tables->n_sources);
        for(int i=0; i < tables->n_sources; ++i) {
            source_hp_pairs[i] = {tables->healpix_id_s[i], i};
        }

        // 2. Sort these pairs by hp_id.
        std::sort(source_hp_pairs.begin(), source_hp_pairs.end(),
                  [](const auto& a, const auto& b){ return a.first < b.first; });

        // 3. Populate d_all_source_hp_ids_sorted_gpu and d_sorted_source_original_indices_gpu
        std::vector<long> h_all_source_hp_ids_sorted(tables->n_sources);
        std::vector<int> h_sorted_source_original_indices(tables->n_sources);
        for(int i=0; i < tables->n_sources; ++i) {
            h_all_source_hp_ids_sorted[i] = source_hp_pairs[i].first;
            h_sorted_source_original_indices[i] = source_hp_pairs[i].second;
        }
        CUDA_CHECK(hipMalloc(&d_all_source_hp_ids_sorted_gpu, tables->n_sources * sizeof(long)));
        CUDA_CHECK(hipMemcpy(d_all_source_hp_ids_sorted_gpu, h_all_source_hp_ids_sorted.data(), tables->n_sources * sizeof(long), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMalloc(&d_sorted_source_original_indices_gpu, tables->n_sources * sizeof(int)));
        CUDA_CHECK(hipMemcpy(d_sorted_source_original_indices_gpu, h_sorted_source_original_indices.data(), tables->n_sources * sizeof(int), hipMemcpyHostToDevice));


        // 4. Find unique sorted HP IDs and their offsets on host (could be done on GPU with Thrust/CUB)
        std::vector<long> h_unique_source_hp_ids;
        std::vector<int> h_unique_hp_id_offsets_start; // inclusive start index in sorted_hp_ids
        std::vector<int> h_unique_hp_id_offsets_end;   // exclusive end index

        if (tables->n_sources > 0) {
            h_unique_source_hp_ids.push_back(h_all_source_hp_ids_sorted[0]);
            h_unique_hp_id_offsets_start.push_back(0);
            for (int i = 1; i < tables->n_sources; ++i) {
                if (h_all_source_hp_ids_sorted[i] != h_all_source_hp_ids_sorted[i-1]) {
                    h_unique_source_hp_ids.push_back(h_all_source_hp_ids_sorted[i]);
                    h_unique_hp_id_offsets_end.push_back(i); // end for previous
                    h_unique_hp_id_offsets_start.push_back(i); // start for current
                }
            }
            h_unique_hp_id_offsets_end.push_back(tables->n_sources); // end for the last unique ID
        }
        N_unique_source_hp = h_unique_source_hp_ids.size();

        if (N_unique_source_hp > 0) {
            CUDA_CHECK(hipMalloc(&d_unique_source_hp_ids, N_unique_source_hp * sizeof(long)));
            CUDA_CHECK(hipMemcpy(d_unique_source_hp_ids, h_unique_source_hp_ids.data(), N_unique_source_hp * sizeof(long), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_unique_hp_id_offsets_start_gpu, N_unique_source_hp * sizeof(int)));
            CUDA_CHECK(hipMemcpy(d_unique_hp_id_offsets_start_gpu, h_unique_hp_id_offsets_start.data(), N_unique_source_hp * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMalloc(&d_unique_hp_id_offsets_end_gpu, N_unique_source_hp * sizeof(int)));
            CUDA_CHECK(hipMemcpy(d_unique_hp_id_offsets_end_gpu, h_unique_hp_id_offsets_end.data(), N_unique_source_hp * sizeof(int), hipMemcpyHostToDevice));

            // 5. Convert unique HEALPix IDs to Cartesian coordinates for KD-Tree construction
            std::vector<float3> h_unique_source_hp_coords_kdtree(N_unique_source_hp);

            for(int i=0; i < N_unique_source_hp; ++i) {
                double theta, phi;
                // Use host version of pix2ang_ring function
                pix2ang_ring_host(tables->nside_healpix, h_unique_source_hp_ids[i], theta, phi);
                
                // Convert angles to Cartesian coordinates
                double sin_theta = sin(theta);
                h_unique_source_hp_coords_kdtree[i].x = static_cast<float>(sin_theta * cos(phi));
                h_unique_source_hp_coords_kdtree[i].y = static_cast<float>(sin_theta * sin(phi));
                h_unique_source_hp_coords_kdtree[i].z = static_cast<float>(cos(theta));
            }
            CUDA_CHECK(hipMalloc(&d_unique_source_hp_coords_kdtree, N_unique_source_hp * sizeof(float3)));
            CUDA_CHECK(hipMemcpy(d_unique_source_hp_coords_kdtree, h_unique_source_hp_coords_kdtree.data(), N_unique_source_hp * sizeof(float3), hipMemcpyHostToDevice));

            // 6. Build KD-Tree using cudaKDTree library
            std::cout << "Building cudaKDTree with " << N_unique_source_hp << " points..." << std::endl;
            
            // Note: We're not using the KD-tree for search anymore due to issues
            // Just keeping the coordinate array for brute force search
            
            std::cout << "KD-Tree built successfully." << std::endl;
        }
    }


    // --- 5. Launch Main Kernel ---
    if (tables->n_lenses > 0 && tables->n_bins > 0 && tables->n_sources > 0 && N_unique_source_hp > 0) {
        int threadsPerBlock = 128; // Or other tuned value
        int blocksPerGrid = (tables->n_lenses + threadsPerBlock - 1) / threadsPerBlock;

        process_all_lenses_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_z_l, d_d_com_l, d_sin_ra_l, d_cos_ra_l, d_sin_dec_l, d_cos_dec_l, d_dist_3d_sq_bins,
            d_z_s, d_d_com_s, d_sin_ra_s, d_cos_ra_s, d_sin_dec_s, d_cos_dec_s,
            d_w_s, d_e_1_s, d_e_2_s, d_z_l_max_s,
            d_unique_source_hp_coords_kdtree, d_unique_source_hp_ids, N_unique_source_hp,
            d_all_source_hp_ids_sorted_gpu, d_sorted_source_original_indices_gpu,
            d_unique_hp_id_offsets_start_gpu, d_unique_hp_id_offsets_end_gpu,
            tables->has_sigma_crit_eff, tables->n_z_bins_l, d_sigma_crit_eff_l, d_z_bin_s,
            tables->has_m_s, d_m_s,
            tables->has_e_rms_s, d_e_rms_s,
            tables->has_R_2_s, d_R_2_s,
            tables->has_R_matrix_s, d_R_11_s, d_R_12_s, d_R_21_s, d_R_22_s,
            tables->n_lenses, tables->n_bins, tables->nside_healpix,
            tables->comoving, tables->weighting,
            d_sum_1_r, d_sum_w_ls_r, d_sum_w_ls_e_t_r, d_sum_w_ls_e_t_sigma_crit_r,
            d_sum_w_ls_z_s_r, d_sum_w_ls_sigma_crit_r,
            d_sum_w_ls_m_r, d_sum_w_ls_1_minus_e_rms_sq_r,
            d_sum_w_ls_A_p_R_2_r, d_sum_w_ls_R_T_r
        );
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    } else {
        std::cout << "Skipping kernel launch due to no lenses, sources, bins, or unique source HP cells." << std::endl;
    }

    // --- 6. Copy Results (Device to Host) ---
    if (total_output_bins > 0) {
        CUDA_CHECK(hipMemcpy(tables->sum_1_r, d_sum_1_r, total_output_bins * sizeof(long long), hipMemcpyDeviceToHost));
        // ... (repeat for all output sum arrays)
        CUDA_CHECK(hipMemcpy(tables->sum_w_ls_r, d_sum_w_ls_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(tables->sum_w_ls_e_t_r, d_sum_w_ls_e_t_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(tables->sum_w_ls_e_t_sigma_crit_r, d_sum_w_ls_e_t_sigma_crit_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(tables->sum_w_ls_z_s_r, d_sum_w_ls_z_s_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(tables->sum_w_ls_sigma_crit_r, d_sum_w_ls_sigma_crit_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));

        if (tables->has_m_s && tables->sum_w_ls_m_r) CUDA_CHECK(hipMemcpy(tables->sum_w_ls_m_r, d_sum_w_ls_m_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        if (tables->has_e_rms_s && tables->sum_w_ls_1_minus_e_rms_sq_r) CUDA_CHECK(hipMemcpy(tables->sum_w_ls_1_minus_e_rms_sq_r, d_sum_w_ls_1_minus_e_rms_sq_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        if (tables->has_R_2_s && tables->sum_w_ls_A_p_R_2_r) CUDA_CHECK(hipMemcpy(tables->sum_w_ls_A_p_R_2_r, d_sum_w_ls_A_p_R_2_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
        if (tables->has_R_matrix_s && tables->sum_w_ls_R_T_r) CUDA_CHECK(hipMemcpy(tables->sum_w_ls_R_T_r, d_sum_w_ls_R_T_r, total_output_bins * sizeof(double), hipMemcpyDeviceToHost));
    }

    // --- 7. Final Cleanup (Free GPU memory) ---
    CUDA_CHECK(hipFree(d_z_l)); CUDA_CHECK(hipFree(d_d_com_l)); /* ... free all d_lens arrays ... */
    CUDA_CHECK(hipFree(d_sin_ra_l)); CUDA_CHECK(hipFree(d_cos_ra_l)); CUDA_CHECK(hipFree(d_sin_dec_l)); CUDA_CHECK(hipFree(d_cos_dec_l));
    CUDA_CHECK(hipFree(d_dist_3d_sq_bins));

    if (tables->n_sources > 0) {
        CUDA_CHECK(hipFree(d_z_s)); CUDA_CHECK(hipFree(d_d_com_s)); /* ... free all d_source arrays ... */
        CUDA_CHECK(hipFree(d_sin_ra_s)); CUDA_CHECK(hipFree(d_cos_ra_s)); CUDA_CHECK(hipFree(d_sin_dec_s)); CUDA_CHECK(hipFree(d_cos_dec_s));
        CUDA_CHECK(hipFree(d_w_s)); CUDA_CHECK(hipFree(d_e_1_s)); CUDA_CHECK(hipFree(d_e_2_s)); CUDA_CHECK(hipFree(d_z_l_max_s));
        CUDA_CHECK(hipFree(d_all_source_hp_ids)); // This was the original unsorted one

        if (tables->has_sigma_crit_eff) { CUDA_CHECK(hipFree(d_sigma_crit_eff_l)); CUDA_CHECK(hipFree(d_z_bin_s)); }
        if (tables->has_m_s) CUDA_CHECK(hipFree(d_m_s));
        if (tables->has_e_rms_s) CUDA_CHECK(hipFree(d_e_rms_s));
        if (tables->has_R_2_s) CUDA_CHECK(hipFree(d_R_2_s));
        if (tables->has_R_matrix_s) { CUDA_CHECK(hipFree(d_R_11_s)); CUDA_CHECK(hipFree(d_R_12_s)); CUDA_CHECK(hipFree(d_R_21_s)); CUDA_CHECK(hipFree(d_R_22_s));}

        // Free KD-tree related GPU arrays
        if(d_unique_source_hp_coords_kdtree) CUDA_CHECK(hipFree(d_unique_source_hp_coords_kdtree));
        if(d_unique_source_hp_ids) CUDA_CHECK(hipFree(d_unique_source_hp_ids));
        // if(d_source_idx_map_from_kdtree_node) CUDA_CHECK(hipFree(d_source_idx_map_from_kdtree_node));
        if(d_all_source_hp_ids_sorted_gpu) CUDA_CHECK(hipFree(d_all_source_hp_ids_sorted_gpu));
        if(d_sorted_source_original_indices_gpu) CUDA_CHECK(hipFree(d_sorted_source_original_indices_gpu));
        if(d_unique_hp_id_offsets_start_gpu) CUDA_CHECK(hipFree(d_unique_hp_id_offsets_start_gpu));
        if(d_unique_hp_id_offsets_end_gpu) CUDA_CHECK(hipFree(d_unique_hp_id_offsets_end_gpu));
    }


    if (total_output_bins > 0) {
        CUDA_CHECK(hipFree(d_sum_1_r)); /* ... free all d_sum output arrays ... */
        CUDA_CHECK(hipFree(d_sum_w_ls_r)); CUDA_CHECK(hipFree(d_sum_w_ls_e_t_r)); CUDA_CHECK(hipFree(d_sum_w_ls_e_t_sigma_crit_r));
        CUDA_CHECK(hipFree(d_sum_w_ls_z_s_r)); CUDA_CHECK(hipFree(d_sum_w_ls_sigma_crit_r));
        if (tables->has_m_s && tables->sum_w_ls_m_r) CUDA_CHECK(hipFree(d_sum_w_ls_m_r));
        if (tables->has_e_rms_s && tables->sum_w_ls_1_minus_e_rms_sq_r) CUDA_CHECK(hipFree(d_sum_w_ls_1_minus_e_rms_sq_r));
        if (tables->has_R_2_s && tables->sum_w_ls_A_p_R_2_r) CUDA_CHECK(hipFree(d_sum_w_ls_A_p_R_2_r));
        if (tables->has_R_matrix_s && tables->sum_w_ls_R_T_r) CUDA_CHECK(hipFree(d_sum_w_ls_R_T_r));
    }

    std::cout << "precompute_cuda_interface completed successfully." << std::endl;
    return 0; // Success
}

// Placeholder for the actual `precompute_engine_cuda.cu` content or where physics kernels are.
// The actual physics functions (dist_3d_sq_kernel, find_bin_idx_kernel, etc.)
// would need to be adapted to work with the global pointers and indices used in the callback.
// For example:
/*
__device__ double calculate_sigma_crit_inv_gpu_global(
    double zl, double zs, double dcoml, double dcoms, bool comoving)
{
    if (zl >= zs) return 0.0; // Invalid configuration

    double d_ls; // Effective distance between lens and source
    if (comoving) {
        d_ls = dcoms - dcoml;
        // Ensure d_ls is not negative due to precision with dcoms ~ dcoml
        if (d_ls < 0) d_ls = 0;
    } else { // Physical distances
        // This case is more complex if dcoml, dcoms are comoving distances
        // Typically, D_LS = D_S - D_L for physical angular diameter distances in flat LCDM
        // Or using (1+z) factors: D_LS = ( D_S - D_L/(1+z_L) ) / (1+z_S) (approx, depends on exact defs)
        // For now, assume inputs are appropriate or this branch is not used with comoving dcoml/dcoms.
        // A common simplification if dcoml, dcoms are comoving:
        d_ls = (dcoms - dcoml) / (1.0 + zs); // This is one form of angular diameter distance D_LS.
                                          // The exact formula depends on curvature and definitions.
                                          // In flat LCDM, D_A(z1,z2) = ( D_M(z2) - D_M(z1) ) / (1+z2)
                                          // So if dcoms = D_M(zs) and dcoml = D_M(zl), then d_ls = (dcoms-dcoml)/(1+zs)
         if (d_ls < 0) d_ls = 0;
    }

    // Sigma_crit_inv = (4 * PI * G / c^2) * (D_L * D_LS / D_S)
    // Constants: 4*PI*G/c^2 in appropriate units (e.g., pc/M_solar)
    // D_L, D_S, D_LS are angular diameter distances.
    // If dcoml, dcoms are comoving transverse distances:
    // D_L = dcoml / (1+zl)
    // D_S = dcoms / (1+zs)
    // D_LS needs care. If using the D_A(z1,z2) form above: D_LS = (dcoms-dcoml)/(1+zs)
    // So, Sigma_crit_inv proportional to (dcoml/(1+zl)) * ((dcoms-dcoml)/(1+zs)) / (dcoms/(1+zs))
    // Sigma_crit_inv proportional to dcoml * (dcoms-dcoml) / (dcoms * (1+zl))

    if (dcoms <= 0) return 0.0; // Source is at observer or behind

    // Using the common formula for Sigma_crit_inv with comoving distances D_L, D_S, D_LS:
    // D_L = dcoml, D_S = dcoms, D_LS = dcoms - dcoml (if comoving=true)
    // D_L = dcoml/(1+zl), D_S = dcoms/(1+zs), D_LS = (dcoms-dcoml)/(1+zs) (if comoving=false, approx for flat LCDM)
    double d_l_ang, d_s_ang, d_ls_ang;
    if (comoving) { // Distances are comoving transverse
        d_l_ang = dcoml;
        d_s_ang = dcoms;
        d_ls_ang = dcoms - dcoml;
        if (d_ls_ang < 0) d_ls_ang = 0;
    } else { // Distances are physical angular diameter
        d_l_ang = dcoml / (1.0 + zl);
        d_s_ang = dcoms / (1.0 + zs);
        // D_A(z_l, z_s)
        d_ls_ang = (dcoms - dcoml) / (1.0 + zs); // Assuming dcoml, dcoms are comoving here.
                                                // This requires clarification if input dcoml/dcoms can be physical.
                                                // If inputs are already D_A, then D_LS is more complex.
        if (d_ls_ang < 0) d_ls_ang = 0;
    }

    if (d_s_ang <= 1e-9) return 0.0; // Avoid division by zero if source is at observer

    // CONST_CRIT is 4*pi*G/c^2 in units matching distances (e.g. Mpc/Msun)
    // Assuming it's defined elsewhere (e.g. precompute_engine_cuda.h)
    // #define CONST_CRIT (4.30091e-9) // Mpc / M_sun, if G in (km/s)^2 Mpc/Msun, c in km/s
    // For now, use 1.0 as placeholder for 4*pi*G/c^2
    return (d_l_ang * d_ls_ang / d_s_ang) * 1.0; // Factor for 4*pi*G/c^2 is missing
}
*/
